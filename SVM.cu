#include "hip/hip_runtime.h"
//
// Created by svmfan on 3/25/17.
//

#include "SVM.h"
#include <iostream>
#include <cstring>

using namespace std;




/*__device__ float kernel_func(float* x, float* b, unsigned int size) {
    float s = 0;
    while( size > 0 ) {
        size--;
        s += x[size] * b[size];
    }

    return s;
}*/

__global__ void fit_kernel() {/*unsigned int* indices, unsigned int T, unsigned int* a, float* x,
                      unsigned int weight_size, float* y, unsigned int data_size, float h) {
    /*cuPrintf("here I am");
    unsigned int id = threadIdx.x;
    unsigned int t =  id + 1;
    unsigned int i = indices[id];

    float q = 1/(h*t);
    float s = 0;

    for(unsigned int j = 0; j < data_size; j++)
        if( j != i ) {
            double k_result = 0;
            for( int i = 0; i < weight_size; i++)
                k_result += x[i * weight_size + i] * x[j * weight_size + i];
            s += a[j] * y[j] * k_result;
        }

    if (y[i] * q * s < 1) {
        atomicAdd(&a[i], 1);
    }*/
}

float dot(float* x, float* b, unsigned int size) {
    float s = 0;
    while( size > 0 ) {
        size--;
        s += x[size] * b[size];
    }

    return s;
}

void produce_vector(float* v, unsigned int size, float h) {
    while( size > 0 ) {
        size--;
        v[size] *= h;
    }
}

SVM::SVM() {
    v = NULL;
    w = NULL;
    x = NULL;
}

void int_vector_to_float(uint8_t* int_x, float* new_x, unsigned int size) {
    while( size > 0 ) {
        size--;
        new_x[size] = int_x[size];
    }
}


void add_to_vector(float* dest, float* source, unsigned int size) {
    while( size > 0 ) {
        size--;
        dest[size] += source[size];
    }
}


void SVM::free_memory() {
    if (v != NULL) {
        free(v);
        v = NULL;
    }
    if (x != NULL) {
        free(x);
        v = NULL;
    }
}

SVM::~SVM() {
    free_memory();
}

void SVM::set(float* x, unsigned int weight_size, unsigned int data_size) {
    this->data_size = data_size;
    this->weight_size = weight_size;
    this->x = x;
}




int SVM::fit(float* x, unsigned int weight_size, float* y, unsigned int data_size, float h, unsigned int T) {
    free_memory();

    set(x, weight_size, data_size);


    unsigned int* a = (unsigned int*)calloc(data_size, sizeof(unsigned int));
    unsigned int indices[T];
    for(unsigned int t = 0; t < T; t++)
        indices[t] = rand() % data_size;


    // copying to device memory
    unsigned int* d_indices;
    unsigned int indices_size = sizeof(unsigned int) * T;
    hipMalloc(&d_indices, indices_size);
    hipMemcpy(d_indices, indices, indices_size, hipMemcpyHostToDevice);

    unsigned int* d_a;
    unsigned int a_size = sizeof(unsigned int) * data_size;
    hipMalloc(&d_a, a_size);
    hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);

    float* d_x;
    unsigned int x_size = sizeof(float) * weight_size * data_size;
    hipMalloc(&d_x, x_size);
    hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice);

    float* d_y;
    unsigned int y_size = sizeof(float) * data_size;
    hipMalloc(&d_y, y_size);
    hipMemcpy(d_y, y, y_size, hipMemcpyHostToDevice);
    cout << "Starting kernel " << endl;
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err) {
        cout << err<< endl;
        exit(-1);
    }
    fit_kernel<<<1, 1>>>();//d_indices, T, d_a, d_x, weight_size, d_y, data_size, h);

    cout << "Finish kernel " << endl;
    hipMemcpy(a, d_a, a_size, hipMemcpyDeviceToHost);

    float q = 1/(h*T);
    v = (float*)malloc(sizeof(float) * data_size);
    for(unsigned int j = 0; j < data_size; j++) {
        v[j] = q * a[j] * y[j];
    }

    w = (float*) calloc(weight_size, sizeof(float));
    float cur_x[weight_size];
    for(unsigned int i = 0; i < data_size; i++) {
        memcpy(cur_x, &x[i * weight_size], weight_size * sizeof(float));
        produce_vector(cur_x, weight_size, v[i]);
        add_to_vector(w, cur_x, weight_size);
    }
    hipFree(d_a);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_indices);
    free(a);
    return 0;
}



float SVM::predict(float* x) {
    float res = 0;
    //for(int i = 0; i < data_size; i++)
      //  res += v[i] * this->kernel(x, this->x[i], weight_size);
    res = dot(x, w, weight_size);
    return res;
}