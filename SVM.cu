#include "hip/hip_runtime.h"
//
// Created by svmfan on 3/25/17.
//

#include "SVM.h"
#include <iostream>
#include <cstring>
#include <cstdlib>
#include <cstdio>

using namespace std;


void checkError() {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err) {
        cout << err<< endl;
        exit(-1);
    }
}

#include <cmath>


__host__ __device__ float kernel_func(float* x, float* b, unsigned int size) {
    float s = 0;
    while( size > 0 ) {
        size--;
        s += (x[size] - b[size])*(x[size] - b[size]);
    }

    return exp(-0.02 * s);
}

__global__ void fit_kernel(unsigned int id_offset,
                           unsigned int* indices, unsigned int T, unsigned int* a, float* x,
                      unsigned int weight_size, float* y, unsigned int data_size, float h, float* kernel_values) {
    unsigned int id = blockIdx.x + id_offset;
    if (id >= T)
        return;

    unsigned int t =  id + 1;
    unsigned int i = indices[id];

    float q = 1/(h*t);
    float s = 0;

    for(unsigned int j = 0; j < data_size; j++)
        if( j != i ) {
            s += a[j] * y[j] * kernel_values[id * data_size + j];
        }

    if (y[i] * q * s < 1) {
        atomicAdd(&a[i], 1);
    }
}

float dot(float* x, float* b, unsigned int size) {
    float s = 0;
    while( size > 0 ) {
        size--;
        s += x[size] * b[size];
    }

    return s;
}

void produce_vector(float* v, unsigned int size, float h) {
    while( size > 0 ) {
        size--;
        v[size] *= h;
    }
}

unsigned int* SVM::d_a = NULL;
float* SVM::d_x = NULL;
float* SVM::d_y = NULL;

void SVM::loadCuda(unsigned int max_data_size, unsigned int weight_size) {
    unsigned int a_size = sizeof(unsigned int) * max_data_size;
    hipMalloc(&d_a, a_size);

    unsigned int x_size = sizeof(float) * weight_size * max_data_size;
    hipMalloc(&d_x, x_size);

    unsigned int y_size = sizeof(float) * max_data_size;
    hipMalloc(&d_y, y_size);
}

SVM::SVM() {
    v = NULL;
    w = NULL;
    x = NULL;

}

void int_vector_to_float(uint8_t* int_x, float* new_x, unsigned int size) {
    while( size > 0 ) {
        size--;
        new_x[size] = int_x[size];
    }
}


void add_to_vector(float* dest, float* source, unsigned int size) {
    while( size > 0 ) {
        size--;
        dest[size] += source[size];
    }
}


void SVM::free_memory() {
    if (v != NULL) {
        free(v);
        v = NULL;
    }
    if (x != NULL) {
        free(x);
        v = NULL;
    }
}

SVM::~SVM() {
    free_memory();

}

void SVM::set(float* x, unsigned int weight_size, unsigned int data_size) {
    this->data_size = data_size;
    this->weight_size = weight_size;
    this->x = x;
}

__global__ void kernel_values_compute(unsigned int t_offset, unsigned int data_size_offset,
                                      unsigned int* indices, float* kernel_values, float* x,
                                      unsigned int weight_size,
                                      unsigned int data_size, unsigned int T) {
    unsigned int t = blockIdx.x + t_offset;
    indices[t]++;
    if( t >= T )
        return;
    unsigned int j = threadIdx.x + data_size_offset;
    if( j >= data_size )
        return;

    unsigned int i = indices[t];
    kernel_values[t * data_size + j] = kernel_func(&x[i * weight_size], &x[j * weight_size], weight_size);
}

float* load_kernel_values(unsigned int* d_indices, unsigned int T, float* d_x, unsigned int weight_size, unsigned int data_size) {
    size_t size = sizeof(float) * T * data_size;
    float* d_kernel_values;
    hipMalloc(&d_kernel_values, size);
    unsigned int max_threads = 1024;
    unsigned int max_blocks = 65535;
    unsigned int t_offset = 0;
    // t_offset = how much t we already performed
    while( T > t_offset ) {
        // data_size_offset = how much data we already performed for the current t
        unsigned int data_size_offset = 0;
        while (data_size > data_size_offset) {
            kernel_values_compute <<< max_blocks, max_threads >>> (t_offset, data_size_offset, d_indices, d_kernel_values, d_x, weight_size, data_size, T);
            data_size_offset += max_threads;
        }
        t_offset += max_blocks;
    }



    return d_kernel_values;

}

int SVM::fit(float* x, unsigned int weight_size, float* y, unsigned int data_size, float h, unsigned int T) {
    free_memory();
    //data_size = data_size % 100;

    set(x, weight_size, data_size);


    unsigned int* a;
    hipHostMalloc(&a, data_size*sizeof(unsigned int));

    unsigned int indices[T];
    for(unsigned int t = 0; t < T; t++)
        indices[t] = rand() % data_size;

    size_t total_size = 0;

    // copying to device memory
    unsigned int* d_indices;
    unsigned int indices_size = sizeof(unsigned int) * T;
    total_size += indices_size;
    hipMalloc(&d_indices, indices_size);
    hipMemcpy(d_indices, indices, indices_size, hipMemcpyHostToDevice);

    unsigned int a_size = sizeof(unsigned int) * data_size;
    hipMemset(d_a, 0, a_size);

    unsigned int x_size = sizeof(float) * weight_size * data_size;
    hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice);

    unsigned int y_size = sizeof(float) * data_size;
    hipMemcpy(d_y, y, y_size, hipMemcpyHostToDevice);


    checkError();
    float* d_kernel_values = load_kernel_values(d_indices, T, d_x, weight_size, data_size);
    //cout << total_size << endl;
    hipDeviceSynchronize();
    //cout << "Starting kernel " << endl;

    unsigned int max_threads = 1024;
    unsigned int max_blocks = 65535;
    unsigned int t_offset = 0;
    // t_offset = how much t we already performed
    while( T > t_offset ) {
        // data_size_offset = how much data we already performed for the current t
        checkError();
        fit_kernel<<<max_blocks, 1>>>(t_offset, d_indices, T, d_a, d_x, weight_size, d_y, data_size, h, d_kernel_values);
        t_offset += max_blocks;
    }

    hipDeviceSynchronize();
    //cout << "Finish kernel " << endl;
    cout << endl;
    for(int i = 0; i < data_size; i++)
        cout << a[i];
    hipMemcpy(a, d_a, a_size, hipMemcpyDeviceToHost);
    checkError();
    cout << endl;

    for(int i = 0; i < data_size; i++)
        cout << a[i];
    cout.flush();
    float q = 1/(h*T);
    v = (float*)malloc(sizeof(float) * data_size);
    for(unsigned int j = 0; j < data_size; j++) {
        v[j] = q * a[j] * y[j];
    }

    hipFree(d_indices);
    hipFree(d_kernel_values);
    hipHostFree(a);
    checkError();
    return 0;
}



float SVM::predict(float* x) {
    float res = 0;
    for(int i = 0; i < data_size; i++)
        res += v[i] * kernel_func(x, &(this->x[i * weight_size]), weight_size);
    return res;
}

void SVM::save(const char* filename) {
    FILE* f = fopen(filename, "wb");
    fwrite(&(this->data_size), sizeof(unsigned int), 1, f);
    fwrite(v, sizeof(float) * this->data_size, 1, f);
    fwrite(&(this->weight_size), sizeof(unsigned int), 1, f);
    fwrite(x, sizeof(float) * this->data_size * this->weight_size, 1, f);
    fclose(f);
}

bool SVM::load(const char* filename) {
    FILE* f = NULL;
    if (f = fopen(filename, "rb")) {
        free_memory();

        fread(&(this->data_size), sizeof(unsigned int), 1, f);

        size_t v_size = sizeof(float) * this->data_size;
        v = (float*)malloc(v_size);
        fread(v, v_size, 1, f);

        fread(&(this->weight_size), sizeof(unsigned int), 1, f);

        size_t x_size = v_size * weight_size;
        x = (float*)malloc(x_size);
        fread(x, x_size, 1, f);

        fclose(f);
        return true;
    }
    return false;
}