
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <cstdlib>
#include <cstdio>
//
// Created by svmfan on 3/25/17.
//

void flip(uint8_t* data, unsigned int size) {
    for(unsigned int off = 0; off < size / 2; off++) {
        unsigned int twin = size - off - 1;
        data[off] ^= data[twin];
        data[twin] ^= data[off];
        data[off] ^= data[twin];
    }
}

void fread_uint32_with_flip(uint32_t* i, FILE* f) {
    fread(i, 4, 1, f);
    flip((uint8_t*)i, 4);
}