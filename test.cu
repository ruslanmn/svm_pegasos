
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>


using namespace std;

__global__ void parallelMultiply(float* a, float* b, int n, float* res) {
        int y = threadIdx.x;
		for(int column = 0; column < n; column++) {
			res[y*n+column] = 0;
			for(int i = 0; i < n; i++)
				res[y*n+column] += a[y*n + i] * b[i*n + column];
		}
}


void multiply(float* a, float* b, int n, float* res) {
	for(int y = 0; y < n; y++) {
		for(int column = 0; column < n; column++) {
			res[y*n+column] = 0;
			for(int i = 0; i < n; i++)
				res[y*n+column] += a[y*n + i] * b[i*n + column];
		}
	}
}

void generateMatrix(float* a, int n) {
	for(int i = 0; i < n*n; i++)
		a[i] = rand();
}

int main() {
	srand(time(0));
	int n = 250;
    size_t size = n*n*sizeof(float);
    clock_t begin_time;
    float* ha = (float*) malloc(sizeof(float) * n * n);
    generateMatrix(ha, n);
    float* hb = (float*) malloc(sizeof(float) * n * n);
    generateMatrix(hb, n);
    float* hres = (float*) malloc(sizeof(float) * n * n);

    begin_time = clock();
    multiply(ha, hb, n, hres);
    std::cout << float(clock () - begin_time)/CLOCKS_PER_SEC << endl;

    
    begin_time = clock();
    float* a;
    hipMalloc(&a, size);
    hipMemcpy(a, ha, size, hipMemcpyHostToDevice);
    float* b;
    hipMalloc(&b, size);
    hipMemcpy(b, hb, size, hipMemcpyHostToDevice);
    float* res;
    hipMalloc(&res, size);
    hipMemcpy(res, hres, size, hipMemcpyHostToDevice);

    hipFree(a);
    hipFree(b);
    hipFree(res);
    
    parallelMultiply<<<1, n>>>(a, b, n, res);
    
    std::cout << float(clock () - begin_time)/CLOCKS_PER_SEC << endl;

	return 0;
}

