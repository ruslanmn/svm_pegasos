#include "hip/hip_runtime.h"
#include <iostream>
#include "mnist_data_loader.h"
#include "SVM.h"
#include "mnist_data_classifier.h"
#include <omp.h>


using namespace std;


int print_train_images() {
    MnistDataLoader mld;
    mld.load_mnist_data("/home/kmeansfan/MNIST Data/images.data", "/home/kmeansfan/MNIST Data/labels.data",
                        "/home/kmeansfan/MNIST Data/test-images.data", "/home/kmeansfan/MNIST Data/test-labels.data");
    uint8_t* labels = mld.get_train_labels();
    uint8_t* images = mld.get_train_images();
    int a;
    for(int i = 0; i < mld.get_train_data_size(); i++) {
        for(int y = 0; y < 28; y++) {
            for(int x = 0; x < 28; x++) {
                if (images[i * mld.get_weight_size() + y*28 + x] == 0)
                    cout << " ";
                else
                    cout << "*";
            }
            cout << endl;
        }
        cout << (int)labels[i] << endl;
        cin >> a;

    }
    return 0;
}

int main() {

    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < 1; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }


    srand(time(NULL));

    MnistDataClassifier mdc("/home/kmeansfan/MNIST Data/images.data", "/home/kmeansfan/MNIST Data/labels.data",
                                      "/home/kmeansfan/MNIST Data/test-images.data", "/home/kmeansfan/MNIST Data/test-labels.data",
                            0.1, 10);



    uint8_t* images = mdc.mdl.get_test_images();
    uint8_t* labels = mdc.mdl.get_test_labels();
    unsigned int test_data_size = mdc.mdl.get_test_data_size();
    unsigned int correct_count = 0;
    correct_count = 0;
    for(unsigned int i = 0; i < test_data_size; i++) {
        if( mdc.predict(&images[i * mdc.mdl.get_weight_size()]) == labels[i] )
            correct_count++;
        cout << i << endl;
    }

    cout << correct_count << "/" << test_data_size << endl;


    return 0;
}
/*
int main_opencl_test() {
    cl_platform_id platform_id;
    clGetPlatformIDs(1, &platform_id, NULL);



    const char* source = "__kernel void square(__global float* input, __global float* output, int N)\n"
            "{\n"
            "    int i = get_global_id(0);\n"
            "    if ( i < N )\n"
            "       output[i] = input[i] * input[i];\n"
            "N = 0;\n"
            "}\n";

    // Get the first GPU device associated with the platform
    cl_device_id device_id;
    clGetDeviceIDs(platform_id, CL_DEVICE_TYPE_GPU, 2, &device_id, NULL);

    cl_context context = clCreateContext(NULL, 1, &device_id, NULL, NULL, NULL);

    cl_program program = clCreateProgramWithSource(context, 1, &source, NULL, NULL);
    if( clBuildProgram(program, 0, NULL, NULL, NULL, NULL) != CL_SUCCESS ) {
        char log[999999];
        clGetProgramBuildInfo(program, device_id, CL_PROGRAM_BUILD_LOG, 999999, log, NULL);
        cout << log << endl;
    }

    cl_kernel kernel = clCreateKernel(program, "square", NULL);



    srand(time(NULL));



    unsigned int n = 99999999;
    cl_float* a = (cl_float*) malloc(sizeof(cl_float) * n);
    cl_float* b = (cl_float*) malloc(sizeof(cl_float) * n);

    for(int i = 0; i < n; i++)
        a[i] = rand() % 1000;


    cl_command_queue cmd_queue = clCreateCommandQueue(context, device_id, 0, NULL);
    cl_mem a_buffer = clCreateBuffer(context, CL_MEM_READ_WRITE | CL_MEM_COPY_HOST_PTR, sizeof(cl_float) * n, a, NULL);
    cl_mem b_buffer = clCreateBuffer(context, CL_MEM_READ_WRITE, sizeof(cl_float) * n, NULL, NULL);



    cl_int err;

    clock_t begin;

    unsigned int localWorkSize = 32;
    unsigned int numWorkGroups = (n + localWorkSize - 1) / localWorkSize;
    unsigned int globalWorkSize = numWorkGroups * localWorkSize;


    begin = clock();
    //err = clEnqueueWriteBuffer(cmd_queue, a_buffer, CL_FALSE, 0, n, a, NULL, NULL, NULL);
    err = clSetKernelArg(kernel, 0, sizeof(cl_mem), &a_buffer);
    clSetKernelArg(kernel, 1, sizeof(cl_mem), &b_buffer);
    clSetKernelArg(kernel, 2, sizeof(int), (void *)&n);

    err = clEnqueueNDRangeKernel(cmd_queue, kernel, 1, NULL, &globalWorkSize, &localWorkSize, 0, NULL, NULL);
    err = clFinish(cmd_queue);
    if (err != CL_SUCCESS )
        cout << "BAD" << endl;

    clEnqueueReadBuffer(cmd_queue, b_buffer, CL_TRUE, 0, sizeof(cl_float) * n, b, 0, NULL, NULL);
    cout << (clock() - begin) << endl;

    cout << "b[24] = " << b[n-1] << ", a[24] = " << a[n-1] << endl;

    begin = clock();
  /*  for(int i = 0; i < n; i++)
        b[i] = a[i] * a[i];
    cout << (clock() - begin) << endl;
    cout << "b[24] = " << b[n-1] << ", a[24] = " << a[n-1] << endl;
}*/